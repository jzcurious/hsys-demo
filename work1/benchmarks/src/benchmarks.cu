#include <hip/hip_runtime.h.hpp>
#include <work1/kernels/kernel_vector_add.cuh>
#include <work1/vector.cuh>

#define EIGEN_NO_CUDA

#include <Eigen/Dense>
#include <benchmark/benchmark.h>
#include <cuda_timer.hpp>

static void BM_EigenVectorAddCPU(benchmark::State& state) {
  auto len = state.range(0);

  Eigen::VectorXf a = Eigen::VectorXf(len);
  Eigen::VectorXf b = Eigen::VectorXf(len);
  Eigen::VectorXf result(len);

  for (auto _ : state) {
    result = a + b;  // lazy RHS
    benchmark::DoNotOptimize(result.data());
    benchmark::ClobberMemory();
  }
}

static void BM_CUDAVectorAddGPU(benchmark::State& state) {
  auto size = state.range(0);

  auto a = hsys::Vector<float>(size);
  auto b = hsys::Vector<float>(size);
  auto c = hsys::Vector<float>(size);

  for (auto _ : state) {
    float elapsed_time = 0;

    {
      CUDATimer timer(elapsed_time);
      hsys::kernel_vector_add<<<hip/hip_runtime.h::cover(size, 128), 128>>>(
          c.accessor(), a.accessor(), b.accessor());
    }

    benchmark::DoNotOptimize(elapsed_time);
    benchmark::ClobberMemory();

    state.SetIterationTime(elapsed_time);
  }
}

void* operator new(std::size_t bytes);  // Dumb clangd!

constexpr int multiplier = 8;
constexpr auto range = std::make_pair(8, 1 << 26);
constexpr auto unit = benchmark::kMillisecond;

BENCHMARK(BM_EigenVectorAddCPU)
    ->Name("Eigen Vector Addition (CPU)")
    ->RangeMultiplier(multiplier)
    ->Ranges({range})
    ->Unit(unit)
    ->UseRealTime()
    ->MeasureProcessCPUTime();

BENCHMARK(BM_CUDAVectorAddGPU)
    ->Name("CUDA Vector Addition (GPU)")
    ->RangeMultiplier(multiplier)
    ->Ranges({range})
    ->Unit(unit)
    ->UseManualTime();

BENCHMARK_MAIN();  // NOLINT
